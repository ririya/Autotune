#include "hip/hip_runtime.h"
#include "tmwtypes.h"

// Set isevenX = 1 if kernel is even in X, iseven = 0 if odd.
__global__ void superconv5(float *dout, const float *d, const float *W,
                           const int32_T Nax, const int32_T Nay, const int32_T Ni, 
                           const int32_T Ndx, const int32_T Ndy, const int32_T Ndxh, 
                           const int32_T Ndyh, const int32_T isevenX, const int32_T isevenY) 
{   
    int32_T x = blockIdx.x; // row of output pixel
    int32_T y = blockIdx.y; // column of output pixel
	int32_T zi = blockIdx.z; // Training example number (4th dimension of d array)
    
    int32_T X = gridDim.x; // dout output width in X
    int32_T Y = gridDim.y; // dout output height in Y
    int32_T Zi = gridDim.z; // number of images
    
    int32_T j = threadIdx.x; // Input map number
	int32_T i = threadIdx.y; // Output map number

	int32_T J = blockDim.x; // Number of input maps
	int32_T I = blockDim.y; // Number of input maps

	// center point of kernel starts on boundary of d
	int32_T mx = x - Ndxh + isevenX;
    int32_T my = y - Ndyh + isevenY;
    
	// For a "full" convolution, add if statements to set m,n range to simulate zero padded input
    float res = 0;
	float dpad = 0;
    int32_T m, n; // x, y of W(1,2)
    #pragma unroll 10
    for (m=-Ndxh; m<=Ndxh-isevenX; m++) {
    	#pragma unroll 10
        for (n=-Ndyh; n<=Ndyh-isevenY; n++) {

			// Perform full convolution (set d to zero if kernel point is outside of image)
			if (((mx + m) < 0) || ((my + n) < 0) || ((mx + m) >= Nax) || ((my + n) >= Nay)) {
				dpad = 0;
			}
			else {
				dpad = d[Nax*Nay*I*zi + Nax*Nay*i + Nax*(my + n) + (mx + m)];
			}
            // loop only over 1st and 2nd dimensions
			res += W[Ndx*Ndy*J*i + Ndx*Ndy*j + (n + Ndyh)*Ndx + (m + Ndxh)] * dpad;
        }        
    }
    
    // (Nm-Nk+1, Nm-Nk+1, Nout, Nin, Ni)
    dout[X*Y*J*Zi*i + X*Y*J*zi + X*Y*j + X*y + x] = res;
}	
