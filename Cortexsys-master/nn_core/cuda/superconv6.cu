#include "hip/hip_runtime.h"
#include "tmwtypes.h"

// Set isevenX = 1 if kernel is even in X, iseven = 0 if odd.
__global__ void superconv6(float *W, const float *A, const float *d,
                           const int32_T Nax, const int32_T Nay, const int32_T Ni, const int32_T Nt,
                           const int32_T Ndx, const int32_T Ndy, const int32_T Ndxh, 
                           const int32_T Ndyh, const int32_T isevenX, const int32_T isevenY) 
{   
    int32_T x = blockIdx.x; // row of output pixel
    int32_T y = blockIdx.y; // column of output pixel
	int32_T ni = blockIdx.z % Ni; // Training example number (4th dimension of A array)
    int32_T nt = blockIdx.z / Ni; // Time step (5th dimension of A array)

    int32_T X = gridDim.x; // W output width in X
    int32_T Y = gridDim.y; // W output height in Y
    
    int32_T j = threadIdx.x; // Input map number
	int32_T i = threadIdx.y; // Output map number

	int32_T J = blockDim.x; // Number of input maps
	int32_T I = blockDim.y; // Number of input maps

    int32_T mx = x + Ndxh;
    int32_T my = y + Ndyh;
    
    float res = 0;
    int32_T m, n; // x, y of d(1,2)
    #pragma unroll 10
    for (m=-Ndxh; m<=Ndxh-isevenX; m++) {
    	#pragma unroll 10
        for (n=-Ndyh; n<=Ndyh-isevenY; n++) {
            // loop only over 1st and 2nd dimensions
            res += d[Ndx*Ndy*I*Ni*nt + Ndx*Ndy*I*ni + Ndx*Ndy*i + (n+Ndyh)*Ndx + (m+Ndxh)] * 
                   A[Nax*Nay*J*Ni*nt + Nax*Nay*J*ni + Nax*Nay*j + Nax*(my+n) + (mx+m)];
        }        
    }
    
    // (Nm-Nk+1, Nm-Nk+1, Nout, Nin, Ni, Nt)
    W[X*Y*J*I*Ni*nt + X*Y*J*I*ni + X*Y*J*i + X*Y*j + X*y + x] = res;
}	
