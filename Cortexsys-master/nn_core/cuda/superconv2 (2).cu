#include "hip/hip_runtime.h"
#include "tmwtypes.h"

// Note: on GTX Titan Black, optimal number of threads/images is 64 (Ni = 64).
// Set isevenX = 1 if kernel is even in X, iseven = 0 if odd.
__global__ void superconv2(float *Mout, const float *M, const float *K, 
                           const int32_T Nmx,  const int32_T Nmy,
                           const int32_T Nkx,  const int32_T Nky, 
                           const int32_T Nkxh, const int32_T Nkyh, 
                           const int32_T isevenX, const int32_T isevenY) 
{   
    int32_T x = blockIdx.x; // row of output pixel
    int32_T y = blockIdx.y; // column of output pixel
    
    int32_T X = gridDim.x; // map output width in X
    int32_T Y = gridDim.y; // map output height in Y
    
    //int32_T Zk = blockDim.x; // number of 2D kernels (kernel depth)
    int32_T zk = threadIdx.x; // map number (3rd dimension of M array)

    int32_T mx = x + Nkxh;
    int32_T my = y + Nkyh;
    
    float res = 0;
    int32_T i, j;
    #pragma unroll 10
    for (i=-Nkxh; i<=Nkxh-isevenX; i++) {
    	#pragma unroll 10
        for (j=-Nkyh; j<=Nkyh-isevenY; j++) {
            // loop only over 1st and 2nd dimensions
            res += K[Nkx*Nky*zk + (j+Nkyh)*Nkx + (i+Nkxh)] * 
                   M[Nmx*Nmy*zk + Nmx*(my+j) + (mx+i)];
        }        
    }
    
    // (Nm-Nk+1, Nm-Nk+1, Nkz, Ni)
    Mout[X*Y*zk + X*y + x] = res;
}	
