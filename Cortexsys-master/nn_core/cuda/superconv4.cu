#include "hip/hip_runtime.h"
#include "tmwtypes.h"

// Set isevenX = 1 if kernel is even in X, iseven = 0 if odd.
__global__ void superconv4(float *W, const float *A, const float *d,
                           const int32_T Nax, const int32_T Nay, const int32_T Ni, 
                           const int32_T Ndx, const int32_T Ndy, const int32_T Ndxh, 
                           const int32_T Ndyh, const int32_T isevenX, const int32_T isevenY) 
{   
    int32_T x = blockIdx.x; // row of output pixel
    int32_T y = blockIdx.y; // column of output pixel
	int32_T zi = blockIdx.z; // Training example number (4th dimension of A array)
    
    int32_T X = gridDim.x; // W output width in X
    int32_T Y = gridDim.y; // W output height in Y
    //int32_T Zi = gridDim.z; // number of images
    
    int32_T j = threadIdx.x; // Input map number
	int32_T i = threadIdx.y; // Output map number

	int32_T J = blockDim.x; // Number of input maps
	int32_T I = blockDim.y; // Number of input maps

    int32_T mx = x + Ndxh;
    int32_T my = y + Ndyh;
    
    float res = 0;
    int32_T m, n; // x, y of d(1,2)
    #pragma unroll 10
    for (m=-Ndxh; m<=Ndxh-isevenX; m++) {
    	#pragma unroll 10
        for (n=-Ndyh; n<=Ndyh-isevenY; n++) {
            // loop only over 1st and 2nd dimensions
            res += d[Ndx*Ndy*I*zi + Ndx*Ndy*i + (n+Ndyh)*Ndx + (m+Ndxh)] * 
                   A[Nax*Nay*J*zi + Nax*Nay*j + Nax*(my+n) + (mx+m)];
        }        
    }
    
    // (Nm-Nk+1, Nm-Nk+1, Nout, Nin, Ni)
    W[X*Y*J*I*zi + X*Y*J*i + X*Y*j + X*y + x] = res;
}	
